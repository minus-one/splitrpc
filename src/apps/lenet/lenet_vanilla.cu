#include "hip/hip_runtime.h"
// Copyright (c) Adithya Kumar, The Pennsylvania State University. All rights reserved.
// Licensed under the MIT License.

#include "lenet_vanilla.cuh"

//static inline int8_t get_work_launch_type() {
//  return readEnvInfo<int16_t>("P2P_RPC_WORK_LAUNCH_TYPE", 1);
//}

__device__ __forceinline__ unsigned long long __globaltimer()
{
  unsigned long long globaltimer;
  asm volatile("mov.u64 %0, %globaltimer;" : "=l"(globaltimer));
  return globaltimer;
}

__global__ void fuse_conv2d_kernel0( float* __restrict__ input0,  float* __restrict__ input1,  float* __restrict__ compute) {
   float compute_local[2];
  __shared__ float pad_temp_shared[180];
  __shared__ float input1_shared[500];
  for (int yy_c_init = 0; yy_c_init < 2; ++yy_c_init) {
    compute_local[yy_c_init] = 0.000000e+00f;
  }
  if ((((int)threadIdx.z) * 9) < ((180 - ((int)threadIdx.y)) - ((int)threadIdx.x))) {
    pad_temp_shared[(((((((((int)threadIdx.z) * 9) + ((int)threadIdx.y)) + ((int)threadIdx.x)) / 180) * 180) + ((((((int)threadIdx.z) * 3) + ((int)threadIdx.y)) + ((int)threadIdx.x)) % 6)) + ((((((((int)threadIdx.z) * 9) + ((int)threadIdx.y)) + ((int)threadIdx.x)) / 6) % 30) * 6))] = (((((1 <= (((((((int)threadIdx.z) * 9) + ((int)threadIdx.y)) + ((int)threadIdx.x)) / 6) % 30)) && ((((((((int)threadIdx.z) * 9) + ((int)threadIdx.y)) + ((int)threadIdx.x)) / 6) % 30) < 29)) && ((1 - ((((((int)threadIdx.z) * 3) + ((int)threadIdx.y)) + ((int)threadIdx.x)) % 6)) <= (((int)blockIdx.x) * 2))) && ((((int)blockIdx.x) * 2) < (29 - ((((((int)threadIdx.z) * 3) + ((int)threadIdx.y)) + ((int)threadIdx.x)) % 6)))) ? input0[(((((((int)blockIdx.x) * 2) + (((((((int)threadIdx.z) * 9) + ((int)threadIdx.y)) + ((int)threadIdx.x)) / 180) * 784)) + ((((((int)threadIdx.z) * 3) + ((int)threadIdx.y)) + ((int)threadIdx.x)) % 6)) + ((((((((int)threadIdx.z) * 9) + ((int)threadIdx.y)) + ((int)threadIdx.x)) / 6) % 30) * 28)) - 29)] : 0.000000e+00f);
  }
  if (((int)threadIdx.z) < (20 - (((((int)threadIdx.y) * 2) + ((int)threadIdx.x)) / 25))) {
    input1_shared[((((((int)threadIdx.z) * 25) + ((((((int)threadIdx.y) * 2) + ((int)threadIdx.x)) / 25) * 25)) + (((((int)threadIdx.y) * 2) + ((int)threadIdx.x)) % 5)) + (((((((int)threadIdx.y) * 2) + ((int)threadIdx.x)) / 5) % 5) * 5))] = input1[((((((int)threadIdx.z) * 25) + ((((((int)threadIdx.y) * 2) + ((int)threadIdx.x)) / 25) * 25)) + (((((int)threadIdx.y) * 2) + ((int)threadIdx.x)) % 5)) + (((((((int)threadIdx.y) * 2) + ((int)threadIdx.x)) / 5) % 5) * 5))];
  }
  __syncthreads();
  for (int ry_inner = 0; ry_inner < 5; ++ry_inner) {
    for (int rx_inner = 0; rx_inner < 5; ++rx_inner) {
      for (int yy_c = 0; yy_c < 2; ++yy_c) {
        compute_local[yy_c] = (compute_local[yy_c] + (pad_temp_shared[(((((((int)threadIdx.y) * 12) + ((int)threadIdx.x)) + (ry_inner * 6)) + rx_inner) + (yy_c * 6))] * input1_shared[(((((int)threadIdx.z) * 25) + (ry_inner * 5)) + rx_inner)]));
      }
    }
  }
  for (int yy_inner_inner_inner = 0; yy_inner_inner_inner < 2; ++yy_inner_inner_inner) {
    compute[(((((((int)blockIdx.x) * 2) + (((int)threadIdx.z) * 676)) + (((int)threadIdx.y) * 52)) + ((int)threadIdx.x)) + (yy_inner_inner_inner * 26))] = compute_local[yy_inner_inner_inner];
  }
}

__global__ void fuse_tanh_kernel0( float* __restrict__ tensor,  float* __restrict__ input0) {
  if ((((int)blockIdx.x) * 512) < (13520 - ((int)threadIdx.x))) {
    tensor[((((int)blockIdx.x) * 512) + ((int)threadIdx.x))] = tanhf(input0[((((int)blockIdx.x) * 512) + ((int)threadIdx.x))]);
  }
}

__global__ void fuse_max_pool2d_kernel0( float* __restrict__ input0,  float* __restrict__ tensor) {
   float tensor_local[1];
  tensor_local[0] = -3.402823e+38f;
  for (int rv = 0; rv < 2; ++rv) {
    for (int rv1 = 0; rv1 < 2; ++rv1) {
      if ((((int)blockIdx.x) * 512) < (3380 - ((int)threadIdx.x))) {
        tensor_local[0] = max(tensor_local[0], input0[(((((((((((int)blockIdx.x) * 512) + ((int)threadIdx.x)) / 3380) * 13520) + ((((((int)blockIdx.x) * 5) + ((int)threadIdx.x)) % 13) * 2)) + (((((((int)blockIdx.x) * 512) + ((int)threadIdx.x)) / 13) % 13) * 52)) + (((((((int)blockIdx.x) * 512) + ((int)threadIdx.x)) / 169) % 20) * 676)) + (rv * 26)) + rv1)]);
      }
    }
  }
  if ((((int)blockIdx.x) * 512) < (3380 - ((int)threadIdx.x))) {
    tensor[(((((((((int)blockIdx.x) * 512) + ((int)threadIdx.x)) / 3380) * 3380) + (((((int)blockIdx.x) * 5) + ((int)threadIdx.x)) % 13)) + (((((((int)blockIdx.x) * 512) + ((int)threadIdx.x)) / 13) % 13) * 13)) + (((((((int)blockIdx.x) * 512) + ((int)threadIdx.x)) / 169) % 20) * 169))] = tensor_local[0];
  }
}


__global__ void fuse_conv2d_1_kernel0( float* __restrict__ input0,  float* __restrict__ input1,  float* __restrict__ compute) {
   float compute_local[1];
  __shared__ float pad_temp_shared[150];
  __shared__ float input1_shared[500];
  compute_local[0] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 10; ++rc_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner < 2; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) {
      if ((((int)threadIdx.z) * 3) < (30 - (((((int)threadIdx.y) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 5))) {
        pad_temp_shared[(((((((((int)threadIdx.z) * 3) + (((((int)threadIdx.y) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 5)) / 30) * 150) + ((((((int)threadIdx.z) * 3) + (((((int)threadIdx.y) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 5)) % 15) * 5)) + (((((int)threadIdx.y) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 5)) + (((((((int)threadIdx.z) * 3) + (((((int)threadIdx.y) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 5)) / 15) % 2) * 75))] = (((((1 <= (((((int)threadIdx.z) * 3) + (((((int)threadIdx.y) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 5)) % 15)) && ((((((int)threadIdx.z) * 3) + (((((int)threadIdx.y) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 5)) % 15) < 14)) && ((1 - (((((int)threadIdx.y) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 5)) <= ((int)blockIdx.x))) && (((int)blockIdx.x) < (14 - (((((int)threadIdx.y) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 5)))) ? input0[((((((((int)blockIdx.x) + (rc_outer * 338)) + ((((((int)threadIdx.z) * 3) + (((((int)threadIdx.y) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 5)) / 30) * 3380)) + ((((((int)threadIdx.z) * 3) + (((((int)threadIdx.y) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 5)) % 15) * 13)) + (((((int)threadIdx.y) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 5)) + (((((((int)threadIdx.z) * 3) + (((((int)threadIdx.y) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 5)) / 15) % 2) * 169)) - 14)] : 0.000000e+00f);
      }
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 < 5; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) {
      if (((int)threadIdx.z) < (10 - (((int)threadIdx.y) / 10))) {
        if ((((int)blockIdx.z) * 10) < ((50 - ((int)threadIdx.z)) - (((int)threadIdx.y) / 10))) {
          input1_shared[(((((((int)threadIdx.z) * 50) + ((((int)threadIdx.y) / 10) * 50)) + ((((int)threadIdx.y) % 5) * 5)) + (((((int)threadIdx.y) / 5) % 2) * 25)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1)] = input1[(((((((((int)blockIdx.z) * 5000) + (rc_outer * 50)) + (((int)threadIdx.z) * 500)) + ((((int)threadIdx.y) / 10) * 500)) + ((((int)threadIdx.y) % 5) * 5)) + (((((int)threadIdx.y) / 5) % 2) * 25)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1)];
        }
      }
    }
    __syncthreads();
for (int rc_inner = 0; rc_inner < 2; ++rc_inner) {
      for (int ry_inner = 0; ry_inner < 5; ++ry_inner) {
        for (int rx_inner = 0; rx_inner < 5; ++rx_inner) {
          compute_local[0] = (compute_local[0] + (pad_temp_shared[((((((int)threadIdx.y) * 5) + (rc_inner * 75)) + (ry_inner * 5)) + rx_inner)] * input1_shared[((((((int)threadIdx.z) * 50) + (rc_inner * 25)) + (ry_inner * 5)) + rx_inner)]));
        }
      }
    }
  }
  compute[((((((int)blockIdx.z) * 1210) + ((int)blockIdx.x)) + (((int)threadIdx.z) * 121)) + (((int)threadIdx.y) * 11))] = compute_local[0];
}

__global__ void fuse_tanh_1_kernel0( float* __restrict__ tensor,  float* __restrict__ input0) {
  if ((((int)blockIdx.x) * 512) < (6050 - ((int)threadIdx.x))) {
    tensor[((((int)blockIdx.x) * 512) + ((int)threadIdx.x))] = tanhf(input0[((((int)blockIdx.x) * 512) + ((int)threadIdx.x))]);
  }
}

__global__ void fuse_max_pool2d_1_kernel0( float* __restrict__ input0,  float* __restrict__ tensor) {
   float tensor_local[1];
  tensor_local[0] = -3.402823e+38f;
  for (int rv = 0; rv < 2; ++rv) {
    for (int rv1 = 0; rv1 < 2; ++rv1) {
      if ((((int)blockIdx.x) * 512) < (1250 - ((int)threadIdx.x))) {
        tensor_local[0] = max(tensor_local[0], input0[(((((((((((int)blockIdx.x) * 512) + ((int)threadIdx.x)) / 1250) * 6050) + ((((((int)blockIdx.x) * 2) + ((int)threadIdx.x)) % 5) * 2)) + (((((((int)blockIdx.x) * 512) + ((int)threadIdx.x)) / 5) % 5) * 22)) + (((((((int)blockIdx.x) * 512) + ((int)threadIdx.x)) / 25) % 50) * 121)) + (rv * 11)) + rv1)]);
      }
    }
  }
  if ((((int)blockIdx.x) * 512) < (1250 - ((int)threadIdx.x))) {
    tensor[(((((((((int)blockIdx.x) * 512) + ((int)threadIdx.x)) / 1250) * 1250) + (((((int)blockIdx.x) * 2) + ((int)threadIdx.x)) % 5)) + (((((((int)blockIdx.x) * 512) + ((int)threadIdx.x)) / 5) % 5) * 5)) + (((((((int)blockIdx.x) * 512) + ((int)threadIdx.x)) / 25) % 50) * 25))] = tensor_local[0];
  }
}

__global__ void fuse_flatten_kernel0( float* __restrict__ tensor,  float* __restrict__ input0) {
  if ((((int)blockIdx.x) * 512) < (1250 - ((int)threadIdx.x))) {
    tensor[((((int)blockIdx.x) * 512) + ((int)threadIdx.x))] = input0[(((((((((int)blockIdx.x) * 512) + ((int)threadIdx.x)) / 1250) * 1250) + (((((((int)blockIdx.x) * 512) + ((int)threadIdx.x)) % 1250) / 25) * 25)) + ((((((((int)blockIdx.x) * 512) + ((int)threadIdx.x)) % 1250) / 5) % 5) * 5)) + ((((((int)blockIdx.x) * 512) + ((int)threadIdx.x)) % 1250) % 5))];
  }
}

__global__ void fuse_dense_kernel0( float* __restrict__ input0,  float* __restrict__ input1,  float* __restrict__ compute,  float* __restrict__ input2) {
   float compute_rf[1];
  __shared__ float red_buf0[64];
  __shared__ float compute1[1];
  compute_rf[0] = 0.000000e+00f;
  for (int k_outer = 0; k_outer < 20; ++k_outer) {
    if (((int)threadIdx.x) < (1250 - (k_outer * 64))) {
      compute_rf[0] = (compute_rf[0] + (input0[(((int)threadIdx.x) + (k_outer * 64))] * input1[(((((int)blockIdx.x) * 1250) + ((int)threadIdx.x)) + (k_outer * 64))]));
    }
  }
  ((volatile __shared__ float*)red_buf0)[((int)threadIdx.x)] = compute_rf[0];
  __syncthreads();
  if (((int)threadIdx.x) < 32) {
    ((volatile __shared__ float*)red_buf0)[((int)threadIdx.x)] = (((volatile __shared__ float*)red_buf0)[((int)threadIdx.x)] + ((volatile __shared__ float*)red_buf0)[(32 + ((int)threadIdx.x))]);
  }
  __syncthreads();
  if (((int)threadIdx.x) < 16) {
    ((volatile __shared__ float*)red_buf0)[((int)threadIdx.x)] = (((volatile __shared__ float*)red_buf0)[((int)threadIdx.x)] + ((volatile __shared__ float*)red_buf0)[(16 + ((int)threadIdx.x))]);
    ((volatile __shared__ float*)red_buf0)[((int)threadIdx.x)] = (((volatile __shared__ float*)red_buf0)[((int)threadIdx.x)] + ((volatile __shared__ float*)red_buf0)[(8 + ((int)threadIdx.x))]);
    ((volatile __shared__ float*)red_buf0)[((int)threadIdx.x)] = (((volatile __shared__ float*)red_buf0)[((int)threadIdx.x)] + ((volatile __shared__ float*)red_buf0)[(4 + ((int)threadIdx.x))]);
    ((volatile __shared__ float*)red_buf0)[((int)threadIdx.x)] = (((volatile __shared__ float*)red_buf0)[((int)threadIdx.x)] + ((volatile __shared__ float*)red_buf0)[(2 + ((int)threadIdx.x))]);
    ((volatile __shared__ float*)red_buf0)[((int)threadIdx.x)] = (((volatile __shared__ float*)red_buf0)[((int)threadIdx.x)] + ((volatile __shared__ float*)red_buf0)[(1 + ((int)threadIdx.x))]);
  }
  __syncthreads();
  if (((int)threadIdx.x) == 0) {
    compute1[0] = ((volatile __shared__ float*)red_buf0)[0];
  }
 if (((int)threadIdx.x) == 0) {
    compute[((int)blockIdx.x)] = (compute1[0] + input2[((int)blockIdx.x)]);
  }
}

__global__ void fuse_softmax_kernel0( float* __restrict__ tensor,  float* __restrict__ input0) {
  tensor[0] = -3.402823e+38f;
  for (int k1 = 0; k1 < 10; ++k1) {
    tensor[0] = max(tensor[0], input0[k1]);
  }
}

__global__ void fuse_softmax_kernel1( float* __restrict__ input0,  float* __restrict__ tensor,  float* __restrict__ tensor1) {
   float tensor_rf[1];
  __shared__ float red_buf0[64];
  tensor_rf[0] = 0.000000e+00f;
  if (((int)threadIdx.x) < 10) {
    tensor_rf[0] = (tensor_rf[0] + __expf((input0[((int)threadIdx.x)] - tensor[0])));
  }
  ((volatile __shared__ float*)red_buf0)[((int)threadIdx.x)] = tensor_rf[0];
  __syncthreads();
  if (((int)threadIdx.x) < 32) {
    ((volatile __shared__ float*)red_buf0)[((int)threadIdx.x)] = (((volatile __shared__ float*)red_buf0)[((int)threadIdx.x)] + ((volatile __shared__ float*)red_buf0)[(32 + ((int)threadIdx.x))]);
  }
  __syncthreads();
  if (((int)threadIdx.x) < 16) {
    ((volatile __shared__ float*)red_buf0)[((int)threadIdx.x)] = (((volatile __shared__ float*)red_buf0)[((int)threadIdx.x)] + ((volatile __shared__ float*)red_buf0)[(16 + ((int)threadIdx.x))]);
    ((volatile __shared__ float*)red_buf0)[((int)threadIdx.x)] = (((volatile __shared__ float*)red_buf0)[((int)threadIdx.x)] + ((volatile __shared__ float*)red_buf0)[(8 + ((int)threadIdx.x))]);
    ((volatile __shared__ float*)red_buf0)[((int)threadIdx.x)] = (((volatile __shared__ float*)red_buf0)[((int)threadIdx.x)] + ((volatile __shared__ float*)red_buf0)[(4 + ((int)threadIdx.x))]);
    ((volatile __shared__ float*)red_buf0)[((int)threadIdx.x)] = (((volatile __shared__ float*)red_buf0)[((int)threadIdx.x)] + ((volatile __shared__ float*)red_buf0)[(2 + ((int)threadIdx.x))]);
    ((volatile __shared__ float*)red_buf0)[((int)threadIdx.x)] = (((volatile __shared__ float*)red_buf0)[((int)threadIdx.x)] + ((volatile __shared__ float*)red_buf0)[(1 + ((int)threadIdx.x))]);
  }
  __syncthreads();
  if (((int)threadIdx.x) == 0) {
    tensor1[0] = ((volatile __shared__ float*)red_buf0)[0];
  }
}
__global__ void fuse_softmax_kernel2( float* __restrict__ tensor,  float* __restrict__ input0,  float* __restrict__ tensor1,  float* __restrict__ tensor2) {
  if (((int)threadIdx.x) < 10) {
    tensor[((int)threadIdx.x)] = (__expf((input0[((int)threadIdx.x)] - tensor1[0])) / tensor2[0]);
//	printf("out: %f\n", tensor[((int)threadIdx.x)]);
  }
}

__global__ void convf(volatile char *in_data, float *data, volatile float* oo) {
  int x = (volatile unsigned char)in_data[threadIdx.x] - 127;
  data[threadIdx.x] = __fdividef((float)x, 128);
	//if(threadIdx.x < 2) {
  //  // SOCKPERF HEADER MANAGEMENT
	//	*((volatile unsigned int *)oo + threadIdx.x) = *((volatile unsigned int *)in_data+threadIdx.x);
	//	if(threadIdx.x == 0)
	//			*(volatile int *)&oo[threadIdx.x] = 0x00000000;//*(int *)&oo[threadIdx.x] & 0xFFFFFF00;
////		if(threadIdx.x == 3)
////				*(volatile int *)&oo[threadIdx.x] = 0x00001400;//*(int *)&oo[threadIdx.x] & 0xFFFFFF00;
	//	
	//} else {

	//    int x;
  //  	float o;
	//	int index = 8 + threadIdx.x - 2;
	//    x = (volatile unsigned char)in_data[index] - 127;
  //  	data[threadIdx.x - 2] = __fdividef((float)x, 128);
	////    printf("%f\n",data[threadIdx.x - 4]);
	//}
}

//bool graphCreated = false;
//hipGraph_t graph;
//hipGraphExec_t instance;
//void lenet_graph_launch(float *data,
//    float *conv1_weight,
//    float *conv2_weight,
//    float *fc2_weight,
//    float *fc2_bias,
//    float *o0,
//    float *o1,
//    float *o2,
//    float *o3,
//    float *o4,
//    volatile g_params *call_params,
//    volatile uint32_t *door_bell,
//    hipStream_t stream
//    ) {
//  if(!graphCreated) {
//    printf("Constructing CUDA graph\n");
//    checkCudaErrors(hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal));
//
//    dim3 dimGrid_convf(1,1,1),  dimBlock_convf(784,1,1);
//    dim3 dimGrid_conv2d(13,1,1),  dimBlock_conv2d(2,13,20);
//    dim3 dimGrid_tanh(27,1,1),  dimBlock_tanh(512,1,1);
//    dim3 dimGrid_pool2d(7,1,1),  dimBlock_pool2d(512,1,1);
//    dim3 dimGrid_conv2d1(11,1,5),  dimBlock_conv2d1(1,11,10);
//    dim3 dimGrid_tanh1(12,1,1),  dimBlock_tanh1(512,1,1);
//    dim3 dimGrid_pool2d1(3,1,1),  dimBlock_pool2d1(512,1,1);
//    dim3 dimGrid_flatten(3,1,1),  dimBlock_flatten(512,1,1);
//    dim3 dimGrid_dense(10,1,1),  dimBlock_dense(64,1,1);
//    dim3 dimGrid_softmax0(1,1,1),  dimBlock_softmax0(1,1,1);
//    dim3 dimGrid_softmax1(1,1,1),  dimBlock_softmax1(64,1,1);
//    dim3 dimGrid_softmax2(1,1,1),  dimBlock_softmax2(64,1,1);
//
//    convf<<<dimGrid_convf,dimBlock_convf, 0, stream>>>((volatile char *)call_params->req, data, (volatile float*)call_params->req);
//    fuse_conv2d_kernel0<<<dimGrid_conv2d,dimBlock_conv2d, 0, stream>>>(data, conv1_weight, o1);
//    fuse_tanh_kernel0<<<dimGrid_tanh,dimBlock_tanh, 0, stream>>>(o2,o1);
//    fuse_max_pool2d_kernel0<<<dimGrid_pool2d,dimBlock_pool2d, 0, stream>>>(o2, o1);
//    fuse_conv2d_1_kernel0<<<dimGrid_conv2d1,dimBlock_conv2d1, 0, stream>>>(o1, conv2_weight, o2);
//    fuse_tanh_1_kernel0<<<dimGrid_tanh1,dimBlock_tanh1, 0, stream>>>(o1, o2);
//    fuse_max_pool2d_1_kernel0<<<dimGrid_pool2d1,dimBlock_pool2d1, 0, stream>>>(o1, o2);
//    fuse_flatten_kernel0<<<dimGrid_flatten,dimBlock_flatten, 0, stream>>>(o1, o2);
//    fuse_dense_kernel0<<<dimGrid_dense,dimBlock_dense, 0, stream>>>(o1, fc2_weight, o3, fc2_bias);
//    fuse_softmax_kernel0<<<dimGrid_softmax0,dimBlock_softmax0, 0, stream>>>(o4, o3);
//    fuse_softmax_kernel1<<<dimGrid_softmax1,dimBlock_softmax1, 0, stream>>>(o3, o4, o0);
//    fuse_softmax_kernel2<<<dimGrid_softmax2,dimBlock_softmax2, 0, stream>>>( ((float*)call_params->resp), o3, o4, o0);
//
//    checkCudaErrors(hipStreamEndCapture(stream, &graph));
//    checkCudaErrors(hipGraphInstantiate(&instance, graph, NULL, NULL, 0));
//    graphCreated=true;
//  }
//  checkCudaErrors(hipGraphLaunch(instance, stream));
//  checkCudaErrors(hipStreamSynchronize(stream));
//}

void lenet_kernel_launch(float *data,
    float *conv1_weight,
    float *conv2_weight,
    float *fc2_weight,
    float *fc2_bias,
    float *o0,
    float *o1,
    float *o2,
    float *o3,
    float *o4,
    volatile g_params *call_params,
    volatile uint32_t *door_bell,
    hipStream_t stream
    ) {
  TRACE_PRINTF("LeNet req: %p, resp: %p\n", (void*)call_params->req, (void*)call_params->resp);
  dim3 dimGrid_convf(1,1,1),  dimBlock_convf(784,1,1);
  dim3 dimGrid_conv2d(13,1,1),  dimBlock_conv2d(2,13,20);
  dim3 dimGrid_tanh(27,1,1),  dimBlock_tanh(512,1,1);
  dim3 dimGrid_pool2d(7,1,1),  dimBlock_pool2d(512,1,1);
  dim3 dimGrid_conv2d1(11,1,5),  dimBlock_conv2d1(1,11,10);
  dim3 dimGrid_tanh1(12,1,1),  dimBlock_tanh1(512,1,1);
  dim3 dimGrid_pool2d1(3,1,1),  dimBlock_pool2d1(512,1,1);
  dim3 dimGrid_flatten(3,1,1),  dimBlock_flatten(512,1,1);
  dim3 dimGrid_dense(10,1,1),  dimBlock_dense(64,1,1);
  dim3 dimGrid_softmax0(1,1,1),  dimBlock_softmax0(1,1,1);
  dim3 dimGrid_softmax1(1,1,1),  dimBlock_softmax1(64,1,1);
  dim3 dimGrid_softmax2(1,1,1),  dimBlock_softmax2(64,1,1);

  //unsigned long long pkt_start = __globaltimer();
  convf<<<dimGrid_convf,dimBlock_convf, 0, stream>>>((volatile char *)call_params->req, data, (volatile float*)call_params->req);
  fuse_conv2d_kernel0<<<dimGrid_conv2d,dimBlock_conv2d, 0, stream>>>(data, conv1_weight, o1);
  fuse_tanh_kernel0<<<dimGrid_tanh,dimBlock_tanh, 0, stream>>>(o2,o1);
  fuse_max_pool2d_kernel0<<<dimGrid_pool2d,dimBlock_pool2d, 0, stream>>>(o2, o1);
  fuse_conv2d_1_kernel0<<<dimGrid_conv2d1,dimBlock_conv2d1, 0, stream>>>(o1, conv2_weight, o2);
  fuse_tanh_1_kernel0<<<dimGrid_tanh1,dimBlock_tanh1, 0, stream>>>(o1, o2);
  fuse_max_pool2d_1_kernel0<<<dimGrid_pool2d1,dimBlock_pool2d1, 0, stream>>>(o1, o2);
  fuse_flatten_kernel0<<<dimGrid_flatten,dimBlock_flatten, 0, stream>>>(o1, o2);
  fuse_dense_kernel0<<<dimGrid_dense,dimBlock_dense, 0, stream>>>(o1, fc2_weight, o3, fc2_bias);
  fuse_softmax_kernel0<<<dimGrid_softmax0,dimBlock_softmax0, 0, stream>>>(o4, o3);
  fuse_softmax_kernel1<<<dimGrid_softmax1,dimBlock_softmax1, 0, stream>>>(o3, o4, o0);
  fuse_softmax_kernel2<<<dimGrid_softmax2,dimBlock_softmax2, 0, stream>>>( ((float*)call_params->resp), o3, o4, o0);
  //printf("[globaltimer] Time in GPU %u ns\n", (__globaltimer() - pkt_start));
  //hipStreamSynchronize(stream);
}

__global__ void lenet_dyn_kernel_launch(float *data,
    float *conv1_weight,
    float *conv2_weight,
    float *fc2_weight,
    float *fc2_bias,
    float *o0,
    float *o1,
    float *o2,
    float *o3,
    float *o4,
    char *inp_buf,
    char *out_buf
    ) {
  dim3 dimGrid_convf(1,1,1),  dimBlock_convf(784,1,1);
  dim3 dimGrid_conv2d(13,1,1),  dimBlock_conv2d(2,13,20);
  dim3 dimGrid_tanh(27,1,1),  dimBlock_tanh(512,1,1);
  dim3 dimGrid_pool2d(7,1,1),  dimBlock_pool2d(512,1,1);
  dim3 dimGrid_conv2d1(11,1,5),  dimBlock_conv2d1(1,11,10);
  dim3 dimGrid_tanh1(12,1,1),  dimBlock_tanh1(512,1,1);
  dim3 dimGrid_pool2d1(3,1,1),  dimBlock_pool2d1(512,1,1);
  dim3 dimGrid_flatten(3,1,1),  dimBlock_flatten(512,1,1);
  dim3 dimGrid_dense(10,1,1),  dimBlock_dense(64,1,1);
  dim3 dimGrid_softmax0(1,1,1),  dimBlock_softmax0(1,1,1);
  dim3 dimGrid_softmax1(1,1,1),  dimBlock_softmax1(64,1,1);
  dim3 dimGrid_softmax2(1,1,1),  dimBlock_softmax2(64,1,1);

  //unsigned long long pkt_start = __globaltimer();
  convf<<<dimGrid_convf,dimBlock_convf>>>((volatile char *)inp_buf, data, (volatile float*)inp_buf);
  hipDeviceSynchronize();
  fuse_conv2d_kernel0<<<dimGrid_conv2d,dimBlock_conv2d>>>(data, conv1_weight, o1);
  hipDeviceSynchronize();
  fuse_tanh_kernel0<<<dimGrid_tanh,dimBlock_tanh>>>(o2,o1);
  hipDeviceSynchronize();
  fuse_max_pool2d_kernel0<<<dimGrid_pool2d,dimBlock_pool2d>>>(o2, o1);
  hipDeviceSynchronize();
  fuse_conv2d_1_kernel0<<<dimGrid_conv2d1,dimBlock_conv2d1>>>(o1, conv2_weight, o2);
  hipDeviceSynchronize();
  fuse_tanh_1_kernel0<<<dimGrid_tanh1,dimBlock_tanh1>>>(o1, o2);
  hipDeviceSynchronize();
  fuse_max_pool2d_1_kernel0<<<dimGrid_pool2d1,dimBlock_pool2d1>>>(o1, o2);
  hipDeviceSynchronize();
  fuse_flatten_kernel0<<<dimGrid_flatten,dimBlock_flatten>>>(o1, o2);
  hipDeviceSynchronize();
  fuse_dense_kernel0<<<dimGrid_dense,dimBlock_dense>>>(o1, fc2_weight, o3, fc2_bias);
  hipDeviceSynchronize();
  fuse_softmax_kernel0<<<dimGrid_softmax0,dimBlock_softmax0>>>(o4, o3);
  hipDeviceSynchronize();
  fuse_softmax_kernel1<<<dimGrid_softmax1,dimBlock_softmax1>>>(o3, o4, o0);
  hipDeviceSynchronize();
  fuse_softmax_kernel2<<<dimGrid_softmax2,dimBlock_softmax2>>>( ((float*)out_buf), o3, o4, o0);
  //fuse_softmax_kernel2<<<dimGrid_softmax2,dimBlock_softmax2>>>( ((float*)inp_buf), o3, o4, o0);
  //hipDeviceSynchronize();
  //printf("Time in GPU %u ns\n", (__globaltimer() - pkt_start));
}

__launch_bounds__(1) __global__ void lenet(float *data,
        float *conv1_weight,
        float *conv2_weight,
        float *fc2_weight,
        float *fc2_bias,
        float *o0,
        float *o1,
        float *o2,
        float *o3,
        float *o4,
        volatile g_params *call_params,
        volatile uint32_t *door_bell
        ) {
  dim3 dimGrid_convf(1,1,1),  dimBlock_convf(784,1,1);
  dim3 dimGrid_conv2d(13,1,1),  dimBlock_conv2d(2,13,20);
  dim3 dimGrid_tanh(27,1,1),  dimBlock_tanh(512,1,1);
  dim3 dimGrid_pool2d(7,1,1),  dimBlock_pool2d(512,1,1);
  dim3 dimGrid_conv2d1(11,1,5),  dimBlock_conv2d1(1,11,10);
  dim3 dimGrid_tanh1(12,1,1),  dimBlock_tanh1(512,1,1);
  dim3 dimGrid_pool2d1(3,1,1),  dimBlock_pool2d1(512,1,1);
  dim3 dimGrid_flatten(3,1,1),  dimBlock_flatten(512,1,1);
  dim3 dimGrid_dense(10,1,1),  dimBlock_dense(64,1,1);
  dim3 dimGrid_softmax0(1,1,1),  dimBlock_softmax0(1,1,1);
  dim3 dimGrid_softmax1(1,1,1),  dimBlock_softmax1(64,1,1);
  dim3 dimGrid_softmax2(1,1,1),  dimBlock_softmax2(64,1,1);

  // Get thread ID.
  uint32_t wait_status;

  while(1) {
    while (1) {
      wait_status = ACCESS_ONCE(*(door_bell));
      if(wait_status == 1 || wait_status == 3) {
        break;
      }
    }

    if (wait_status != 1 && wait_status != 2)
      break;

    // Do Work
    //unsigned long long pkt_start = __globaltimer();
    convf<<<dimGrid_convf,dimBlock_convf>>>((volatile char *)call_params->req, data, (volatile float*)call_params->req);
    hipDeviceSynchronize();
    fuse_conv2d_kernel0<<<dimGrid_conv2d,dimBlock_conv2d>>>(data, conv1_weight, o1);
    hipDeviceSynchronize();
    fuse_tanh_kernel0<<<dimGrid_tanh,dimBlock_tanh>>>(o2,o1);
    hipDeviceSynchronize();
    fuse_max_pool2d_kernel0<<<dimGrid_pool2d,dimBlock_pool2d>>>(o2, o1);
    hipDeviceSynchronize();
    fuse_conv2d_1_kernel0<<<dimGrid_conv2d1,dimBlock_conv2d1>>>(o1, conv2_weight, o2);
    hipDeviceSynchronize();
    fuse_tanh_1_kernel0<<<dimGrid_tanh1,dimBlock_tanh1>>>(o1, o2);
    hipDeviceSynchronize();
    fuse_max_pool2d_1_kernel0<<<dimGrid_pool2d1,dimBlock_pool2d1>>>(o1, o2);
    hipDeviceSynchronize();
    fuse_flatten_kernel0<<<dimGrid_flatten,dimBlock_flatten>>>(o1, o2);
    hipDeviceSynchronize();
    fuse_dense_kernel0<<<dimGrid_dense,dimBlock_dense>>>(o1, fc2_weight, o3, fc2_bias);
    hipDeviceSynchronize();
    fuse_softmax_kernel0<<<dimGrid_softmax0,dimBlock_softmax0>>>(o4, o3);
    hipDeviceSynchronize();
    fuse_softmax_kernel1<<<dimGrid_softmax1,dimBlock_softmax1>>>(o3, o4, o0);
    hipDeviceSynchronize();
    fuse_softmax_kernel2<<<dimGrid_softmax2,dimBlock_softmax2>>>( ((float*)call_params->resp), o3, o4, o0);
    hipDeviceSynchronize();
    //printf("[globaltimer based] Time in GPU %u ns\n", (__globaltimer() - pkt_start));

    // Signal work to be complete
    ACCESS_ONCE(*(door_bell)) = 2;
    __threadfence_system();
  }
}

void load_from_file(std::string fname, float* buffer, unsigned int size, unsigned int offset) {
    int fd, ret;
    printf("Loading file: %s\n", fname.c_str());
    fd = open(fname.c_str(), O_RDONLY);
    float *h_buff = new float[size];
    ret = pread(fd, h_buff, size, offset);
    if (ret != size) {
        perror("read error");
        delete[] h_buff;
        exit(-1);
    }
    hipMemcpy(buffer, h_buff, size, hipMemcpyHostToDevice);
    delete[] h_buff;
    close(fd);
}

// Globals
float *o0, *o1, *o2, *o3, *o4, /**oo ,*/ *data, 
      *conv1_weight, *conv2_weight, 
      *fc2_weight, *fc2_bias;

void cuda_init()
{
  hipMalloc(&o0, sizeof(float)* 1024);
  hipMalloc(&o1, sizeof(float)* 13520);
  hipMalloc(&o2, sizeof(float)* 13520);
  hipMalloc(&o3, sizeof(float)* 10);
  hipMalloc(&o4, sizeof(float)* 1024);
  hipMalloc(&data, sizeof(float)* 28 * 28);
  hipMalloc(&conv1_weight, sizeof(float)* 500);
  hipMalloc(&conv2_weight, sizeof(float)* 25000);
  hipMalloc(&fc2_weight, sizeof(float)* 12500);
  hipMalloc(&fc2_bias, sizeof(float)* 10);

  std::string data_set_path = getDatasetBasePath() + std::string("data/lenet/");
  load_from_file(data_set_path + std::string("data2.dat"), conv1_weight,2000,0);
  load_from_file(data_set_path + std::string("data0.dat"), conv2_weight,100000,0);
  load_from_file(data_set_path + std::string("data1.dat"), fc2_weight,50000,0);
  load_from_file(data_set_path + std::string("data3.dat"), fc2_bias,40,0);
}

void kernel_entry(AppCtx *app_ctx) 
{
  lenet_kernel_launch(data, conv1_weight, conv2_weight, fc2_weight, fc2_bias, 
      o0, o1, o2, o3, o4, 
      app_ctx->h_stub, app_ctx->door_bell, app_ctx->work_stream); 
}

void cuda_graph_entry(AppCtx *app_ctx)
{
  if(!app_ctx->graphCreated) {
    printf("Constructing CUDA graph for LeNet\n");
    checkCudaErrors(hipStreamBeginCapture(app_ctx->work_stream, hipStreamCaptureModeGlobal));

    kernel_entry(app_ctx);
    checkCudaErrors(hipStreamEndCapture(app_ctx->work_stream, &app_ctx->graph));
    checkCudaErrors(hipGraphInstantiate(&app_ctx->instance, app_ctx->graph, NULL, NULL, 0));
    app_ctx->graphCreated = true;
  }
  checkCudaErrors(hipGraphLaunch(app_ctx->instance, app_ctx->work_stream));

  //lenet_graph_launch(data, conv1_weight, conv2_weight, fc2_weight, fc2_bias, 
  //    o0, o1, o2, o3, o4, 
  //    app_ctx->h_stub, app_ctx->door_bell, app_ctx->work_stream); 
}

void cdp_entry(AppCtx *app_ctx)
{
  lenet_dyn_kernel_launch<<<1, 1, 0, app_ctx->work_stream>>>(data, conv1_weight, conv2_weight, fc2_weight, fc2_bias, 
      o0, o1, o2, o3, o4, 
      (char*)app_ctx->h_stub->req, (char*)app_ctx->h_stub->resp);
}

void pt_entry(AppCtx *app_ctx)
{
  lenet<<<1, 1, 0, app_ctx->work_stream>>>(data, conv1_weight, conv2_weight, fc2_weight, fc2_bias, 
      o0, o1, o2, o3, o4, 
      app_ctx->d_stub, app_ctx->d_door_bell);
}
